/* 
 * Compile with 
 * nvcc -gencode arch=compute_30,code=sm_30 substringSearchNoStreams.cu
 */



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <limits.h>

#define numStreams 3
#define BLOCK 1024 

__device__ int count_dev;

/*error checking from D&W*/
void errorChecking(hipError_t err, int line) {
    if (err != hipSuccess){
        printf(" %s in %s at line %d\n", hipGetErrorString(err), 
        __FILE__, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void search_kernel(char *string, int length, char *pattern,
        int patternLength) {

    int tx = threadIdx.x;
    int idx = blockDim.x * blockIdx.x + tx; 

    // dynamically allocated shared memory

    extern __shared__ char shared[];
    char *string_sh = &shared[0]; // size BLOCK + patternLength for halo data
    char *pattern_sh = &shared[BLOCK + patternLength]; 

    // These threads load the pattern into shared plus the halo data
    if(tx < patternLength) {
        pattern_sh[tx] = pattern[tx];
        shared[BLOCK + tx] = string[idx + BLOCK];
    }
    string_sh[tx] = string[idx]; 
    __syncthreads();

    // This data can't match since there isn't enough room for the full pattern
    // at the end of the array 
    if(idx >= length - patternLength - 1) {
       return;
    }

    for(int j = 0; j < patternLength; ++j){
       if((pattern_sh[j] ^ string_sh[tx + j]) != 0x0000 ) {
          return; 
       }
    }

    atomicAdd(&count_dev, 1);
}

void search(char * string, int length, char *pattern, int patternLength) {
    char * string_dev, *pattern_dev;
    int count = 0;
    int numThreads = BLOCK;

    dim3 dimGrid( ceil(length/(float)numThreads), 1, 1);
    dim3 dimBlock(numThreads, 1, 1);

    errorChecking( hipMalloc((void **) &string_dev, sizeof(char) * length), 
        __LINE__);
    errorChecking( hipMalloc((void **) &pattern_dev, sizeof(char) * patternLength + 1),
        __LINE__);

    errorChecking( hipMemcpyToSymbol(HIP_SYMBOL(count_dev), &count, 
        sizeof(int), 0, hipMemcpyHostToDevice), __LINE__);
    errorChecking( hipMemcpy(pattern_dev, pattern, 
        patternLength + 1 * sizeof(char), hipMemcpyHostToDevice), __LINE__);   
    errorChecking( hipMemcpy(string_dev, string,  length * sizeof(char), 
        hipMemcpyHostToDevice), __LINE__);
        
    // sharedMem stores the lengths used in the kernel for shared memory
    int sharedMem = (BLOCK+patternLength) * sizeof(char)+patternLength * 
       sizeof(char);
    search_kernel<<<dimGrid.x, dimBlock.x, sharedMem >>>(string_dev, 
        length, pattern, patternLength);

    errorChecking(hipGetLastError(), __LINE__);
    
    errorChecking(hipMemcpyFromSymbol(&count, HIP_SYMBOL(count_dev), 
        sizeof(int), 0, hipMemcpyDeviceToHost), __LINE__);
    
    printf("Count is: %d\n", count);
  
    hipFree(string_dev);
    hipFree(pattern_dev);
}

// Grab data from external file
int get_string_from_file(char *filename, char **input) {
    FILE *file;
    int length;
    size_t result;

    file = fopen(filename, "r");
    fseek(file, 0, SEEK_END);
    length = ftell(file);
    int paddedLength = numStreams * ceil((float)length/numStreams);
    rewind(file);

    errorChecking( hipHostMalloc((void**) input, paddedLength * sizeof(char), hipHostMallocDefault), 
        __LINE__ );
    if(*input == NULL) {
        fputs("Memory error", stderr);
        exit(2);
    }

    result = fread(*input, 1, length, file);
    if(result != length) {
        fputs("Reading error", stderr);
        exit(3);
    }
    fclose(file);
    if(paddedLength > length) {
        for(int i = length - 1; i < paddedLength - 1; i++) {
            (*input)[i] = 0;
        }
        (*input)[paddedLength - 1] = '\0';
    }
    return paddedLength; 
}

// Generate simple string
int generate_string(int length, char **input) {
    int paddedLength = numStreams * ceil((float)length/numStreams);
    errorChecking( hipHostMalloc((void**) input, paddedLength * sizeof(char), hipHostMallocDefault), 
        __LINE__ );
    if(*input == NULL) {
        fputs("Memory error", stderr);
        exit(2);
    }
    for(int i = 0; i < paddedLength - 2; i++) {
        (*input)[i] = 'a';
    }
    (*input)[paddedLength - 1] = '\0';

    int midStream = ceil((float)length/numStreams) - 3;
    printf("Midstream: %d\n", midStream);
    (*input)[midStream] = 'h';
    (*input)[midStream+1] = 'e';
    (*input)[midStream+2] = 'l';
    (*input)[midStream+3] = 'l'; 
    (*input)[midStream+4] = 'o';

    return paddedLength; 
}

int get_pattern(char **pattern) {
    int patternLength = 5;    
    errorChecking( hipHostMalloc((void**) pattern, (patternLength + 1) * sizeof(char), hipHostMallocDefault),
        __LINE__ );
    (*pattern)[0] = 'h';
    (*pattern)[1] = 'e';
    (*pattern)[2] = 'l';
    (*pattern)[3] = 'l';
    (*pattern)[4] = 'o';
    (*pattern)[5] = '\0';

    return patternLength;
}

int main(void) {
    int length = 1024;
    char *string, *pattern;
    int patternLength;
    struct timeval start, end, diff;
    length = get_string_from_file("../DATA/UnicodeSample.txt", &string); 
    patternLength = get_pattern(&pattern);
    printf("Pattern is: %s\n", pattern); 

    gettimeofday(&start, 0); 
    search(string, length, pattern, patternLength);
    gettimeofday(&end, 0); 
    timersub(&start, &end, &diff);
    long long elapsed = (end.tv_sec-start.tv_sec)*1000000ll + end.tv_usec-start.tv_usec;
    printf("GPU Time: %lld \n", elapsed);
    printf("GPU Time (no streams): %ld (msecs) \n", diff.tv_usec);

    hipHostFree(string);
    hipHostFree(pattern);
    return 0;
}

/* SCRATCH CODE
   Compile with nvcc -gencode arch=compute_30,code=sm_30 substringSearch.cu
 */



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <limits.h>

#define numStreams 3
#define BLOCK 256
#define BUFF_SIZE 4096

/*error checking from D&W*/
void errorChecking(hipError_t err, int line) {
   if (err != hipSuccess){
      printf(" %s in %s at line %d\n", hipGetErrorString(err), 
            __FILE__, line);
      exit(EXIT_FAILURE);
   }
}


//THE KERNEL: This, I think, is all the functionality we want out of the kernel.

//Search length must be number of bytes
__global__ void search_kernel(char * patterns_and_output, int pattern_length, char * search_segment, int search_length) {
   int tx = threadIdx.x;
   int idx = blockDim.x * blockIdx.x + tx; 

   int pattern_section = 128;
   int j = 0;

   int count = 0;
   int * result = (int*)(patterns_and_output + idx*sizeof(char)*(pattern_section));
   char * pattern = (char*)(result);

   const char *s;

   for(s = pattern + pattern_section - sizeof(int); !(*s); --s) {}
   pattern_length = (s - pattern) - 4;

   pattern = pattern + 4;

   for(int i = 0; i < search_length - pattern_section; ++i)
   {
      if(pattern[0] == search_segment[i])
      {
         for(j = 0; pattern[j] == search_segment[i + j] && j <= pattern_length && pattern[j]; ++j) {}
         if(j - 1 == pattern_length)
            ++count;
      }
   }

   *result = count;
}

//The following two functions need attribution, since I stole them from K&R
//Utility function for string parsing
void reverse(char s[])
{
   int i, j;
   char c;

   for (i = 0, j = strlen(s)-1; i<j; i++, j--) {
      c = s[i];
      s[i] = s[j];
      s[j] = c;
   }
}

//Utility functionfor string parsing
void itoa(int n, char s[])
{
   int i, sign;

   if ((sign = n) < 0) 
      n = -n;          
   i = 0;
   do { 
      s[i++] = n % 10 + '0';  
   } while ((n /= 10) > 0);  
   if (sign < 0)
      s[i++] = '-';
   s[i] = '\0';
   reverse(s);
}

//THE PRIMARY WORKHORSE
//This function, with these arguments, should be callable in order to start a kernel
//I'm not sure how streams fit into this.  ie. should we allocate a static int in this
//function to determine which stream we kicked off, then increment it and block on it 
//(to prevent too many concurrent streams)?
//int search(char * pattern_, int pattern_length, char * string, int string_length) {
int search(char * keys, int keys_length, int key_count, char * data, int data_length) {
   //These two lines just debug to make sure my formatting is correct.

   //Couldn't get this stuff to work
   char * keys_dev;
   char * data_dev;
   int numThreads = BLOCK;

   dim3 dimGrid( ceil(key_count/(float)numThreads), 1, 1);
   dim3 dimBlock(numThreads, 1, 1);

   printf("dimGrid.x: %d Threads: %d \n" , dimGrid.x, dimBlock.x);

   errorChecking( hipMalloc((void **) &keys_dev, sizeof(char) * keys_length), 
         __LINE__);

   errorChecking( hipMalloc((void **) &data_dev, sizeof(char) * data_length), 
         __LINE__);

      errorChecking( hipMemcpy(keys_dev, 
               keys,  keys_length * sizeof(char), 
               hipMemcpyHostToDevice), __LINE__);

      errorChecking( hipMemcpy(data_dev, 
               data, data_length * sizeof(char), 
               hipMemcpyHostToDevice), __LINE__);

      search_kernel<<<dimGrid.x, dimBlock.x>>>
         (keys_dev, keys_length, data_dev, data_length);

      errorChecking(hipGetLastError(), __LINE__);

      errorChecking(hipMemcpy(keys, keys_dev, keys_length * sizeof(char),
                     hipMemcpyDeviceToHost), __LINE__);

      FILE * outfile = fopen("output.txt", "a");

      char BUFFER[500];
      for(int i = 0; i < key_count; ++i) {
         int num = *(int*)(keys+128*i);
         
         itoa(num, BUFFER);

         fputs(BUFFER, outfile);
         fputs("  -  ", outfile);
         strcpy(BUFFER, (keys + 128*i + sizeof(int)));
         fputs(BUFFER, outfile);
         
         fputs("\n", outfile);
      }    

   fclose(outfile);

   hipFree(data_dev);
   hipFree(keys_dev);

   return 1;
}

// Grab data from external file
int get_string_from_file(char *filename, char **input) {
   FILE *file;
   int length;
   size_t result;

   file = fopen(filename, "r");
   fseek(file, 0, SEEK_END);
   length = ftell(file);
   printf("File Length is: %d bytes.\n", length);
   rewind(file);

   errorChecking( hipHostMalloc((void**) input, length * sizeof(char), hipHostMallocDefault), 
         __LINE__ );
   if(*input == NULL) {
      fputs("Memory error", stderr);
      exit(2);
   }

   result = fread(*input, 1, length, file);
   if(result != length) {
      fputs("Reading error", stderr);
      exit(3);
   }
   fclose(file);

   return length;
}

//This (rather messy) function grabs keys from the file specified in main and writes
//the key and the int returned by search() to the key_value_pairs file
int create_key_results_string(char *filename, char ** string, int * key_count) 
{
   FILE *keyfile;

   char BUFFER[128];
   int keyfile_length = 0;
   int length = 0;
   int j = 0;
   int i = 0;

   for (int i = 0; i < 128; ++i)
      BUFFER[i] = '\0';

   keyfile = fopen(filename, "r");
   if(!keyfile)
      printf("keyfile didn't open");

   fseek(keyfile, 0, SEEK_END);
   keyfile_length = ftell(keyfile);
   rewind(keyfile);

   while(fgets(BUFFER, 128, keyfile))
   {  
      length = strlen(BUFFER) - 1;
      if(BUFFER[length] == '\n')
         ++i;
   }

   *key_count = i;

   errorChecking( hipHostMalloc((void**) string, 128 * i * sizeof(char), hipHostMallocDefault), 
         __LINE__ );
   if(*string == NULL) {
      fputs("Memory error", stderr);
      exit(2);
   }
   rewind(keyfile);

   for(i = 0; fgets(BUFFER, 128, keyfile) != NULL; ++i)
   {
      length = strlen(BUFFER) - 1;
      if(BUFFER[length] == '\n')
         BUFFER[length] = '\0';
      else
      {
         --i;
         continue;
      }
      
      for(j = 0; *(BUFFER + j) != '\0' && j < 128; ++j)
      {
         *(*string + 128*i + j) = *(BUFFER + j);
      }

      if(j == 128)
      {
         --i;
         continue;
      }


      for(; j < 128; ++j)
         *(*string + 128*i + j) = '\0';
   }

   return (128 * *key_count);
}

int generate_string(int length, char **input) {
   int paddedLength = numStreams * ceil((float)length/numStreams);
   errorChecking( hipHostMalloc((void**) input, paddedLength * sizeof(char), hipHostMallocDefault), 
         __LINE__ );
   if(*input == NULL) {
      fputs("Memory error", stderr);
      exit(2);
   }
   for(int i = 0; i < paddedLength - 2; i++) {
      (*input)[i] = 'a';
   }
   (*input)[paddedLength - 1] = '\0';
   return paddedLength; 
}

//I think this is what this function's final version should look like
int main(void) {
   int data_length = 0;
   int keys_length = 0;
   int key_count = 0;
   char * data;
   char * keys;
   struct timeval start, end;
   data_length = get_string_from_file("parsedComments.txt", &data); 
   keys_length = create_key_results_string("key_list.txt", &keys, &key_count);

   printf("%s", keys);

   gettimeofday(&start, 0); 
   search(keys, keys_length, key_count, data, data_length);
   gettimeofday(&end, 0); 

   long long elapsed = (end.tv_sec-start.tv_sec)*1000000ll + end.tv_usec-start.tv_usec;
   printf("GPU Time: %lld \n", elapsed);

   hipHostFree(keys);
   hipHostFree(data);
   return 0;
}

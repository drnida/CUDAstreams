/* SCRATCH CODE
Compile with nvcc -gencode arch=compute_30,code=sm_30 substringSearch.cu
*/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <limits.h>

#define numStreams 3
#define BLOCK 256

__device__ int count_dev;

/*error checking from D&W*/
void errorChecking(hipError_t err, int line) {
    if (err != hipSuccess){
        printf(" %s in %s at line %d\n", hipGetErrorString(err), 
        __FILE__, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void search_kernel(char * string, int length, int offset) {
    int tx = threadIdx.x;
    int idx = offset + blockDim.x * blockIdx.x + tx; 
    int match = 0; 

    int patternLength = 5;
    char pattern[6] = "hello";

    // This data can't match since there isn't enough room for the full pattern
    // at the end of the array 
    if(idx >= length - patternLength - 1) {
       return;
    }

    for(int j = 0; j < patternLength; ++j){
       if((pattern[j] ^ string[idx + j]) == 0x0000 ){
          match += 1; 
       }
    }

    if(match == patternLength) {
       atomicAdd(&count_dev,1);
    }
}

void search(char * string, int length) {
    char * string_dev;
    int count = 0;
    int streamOffset;
    hipStream_t stream[numStreams];
    int numThreads = BLOCK;

    for( int i = 0; i < numStreams; ++ i){
        errorChecking( hipStreamCreate(&stream[i] ), __LINE__);
    }
    
    
    int streamLength = ceil((float)length/numStreams);
    int streamBytes = streamLength * sizeof(char);

    printf("streamLength: %d, streamBytes %d\n", streamLength, streamBytes);    

    dim3 dimGrid( ceil(streamLength/(float)numThreads), 1, 1);
    dim3 dimBlock(numThreads, 1, 1);

    printf("dimGrid.x: %d Threads: %d \n" , dimGrid.x, dimBlock.x);

    errorChecking( hipMalloc((void **) &string_dev, sizeof(char) * length), 
       __LINE__);

    errorChecking( hipMemcpyToSymbol(HIP_SYMBOL(count_dev), &count, 
         sizeof(int), 0, hipMemcpyHostToDevice), __LINE__);
    

    for(int i = 0; i < numStreams; ++i){
        streamOffset = i * streamLength;
        printf("streamOffset is: %d\n", streamOffset);

        errorChecking( hipMemcpyAsync(&string_dev[streamOffset], 
            &string[streamOffset],  streamLength * sizeof(char), 
            hipMemcpyHostToDevice, stream[i] ), __LINE__);
    }    
        
    for(int i = 0; i < numStreams; ++i){
        streamOffset = i * streamLength;
        search_kernel<<<dimGrid.x, dimBlock.x, 0, stream[i]>>>(string_dev, 
           length, streamOffset);

        errorChecking(hipGetLastError(), __LINE__);
    }    

    hipStreamSynchronize(stream[2]); 

    errorChecking(hipMemcpyFromSymbol(&count, HIP_SYMBOL(count_dev), 
        sizeof(int), 0, hipMemcpyDeviceToHost), __LINE__);
        
    printf("Count is: %d\n", count);
  
    for(int i = 0; i < numStreams; ++i){ 
        errorChecking(hipStreamDestroy(stream[i]), __LINE__); 
    }
    hipFree(string_dev);
}

// Grab data from external file
int get_string_from_file(char *filename, char **input) {
    FILE *file;
    int length;
    size_t result;

    file = fopen(filename, "r");
    fseek(file, 0, SEEK_END);
    length = ftell(file);
    printf("File Length is: %d bytes.\n", length);
    int paddedLength = numStreams * ceil((float)length/numStreams);
    rewind(file);

    errorChecking( hipHostMalloc((void**) input, paddedLength * sizeof(char), hipHostMallocDefault), 
        __LINE__ );
    if(*input == NULL) {
        fputs("Memory error", stderr);
        exit(2);
    }

    result = fread(*input, 1, length, file);
    if(result != length) {
        fputs("Reading error", stderr);
        exit(3);
    }
    fclose(file);
    if(paddedLength > length) {
        for(int i = length - 1; i < paddedLength - 1; i++) {
            (*input)[i] = 0;
        }
        (*input)[paddedLength - 1] = '\0';
    }
    return paddedLength; 
}

// Generate simple string
int generate_string(int length, char **input) {
    int paddedLength = numStreams * ceil((float)length/numStreams);
    errorChecking( hipHostMalloc((void**) input, paddedLength * sizeof(char), hipHostMallocDefault), 
        __LINE__ );
    if(*input == NULL) {
        fputs("Memory error", stderr);
        exit(2);
    }
    for(int i = 0; i < paddedLength - 2; i++) {
        (*input)[i] = 'a';
    }
    (*input)[paddedLength - 1] = '\0';
    return paddedLength; 
}

int main(void) {
    int length = 1024;
    char * string;
    struct timeval start, end;
    length = get_string_from_file("UnicodeSample.txt", &string); 

    printf("Padded length is: %d bytes.\n", length);

    gettimeofday(&start, 0); 
    search(string, length);
    gettimeofday(&end, 0); 

    long long elapsed = (end.tv_sec-start.tv_sec)*1000000ll + end.tv_usec-start.tv_usec;
    printf("GPU Time: %lld \n", elapsed);


    hipHostFree(string);
    return 0;
}

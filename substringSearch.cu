/* SCRATCH CODE
Compile with nvcc -gencode arch=compute_30,code=sm_30 substringSearch.cu
*/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <limits.h>

#define numStreams 3


__device__ int count_dev;

/*error checking from D&W*/
void errorChecking(hipError_t err, int line) {
    if (err != hipSuccess){
        printf(" %s in %s at line %d\n", hipGetErrorString(err), 
        __FILE__, line);
        exit(EXIT_FAILURE);
    }
}


__global__ void search_kernel(char * string, char * results, int * numbers, int length) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int patternLength = 5;
    char pattern [6] = "hello";
    int match = 1;
    if(i < length - (patternLength-1)){
        for(int j = 0; j < 5; ++j){
           if((pattern[j] ^ string[i + j]) != 0x0000 ){
            //if( (pattern[j] == string[i + j]) ){
              match = 0; 
           //   results[j] = string[i + j];
            }
        }
    }
    else {
        match =0;
    }
    __syncthreads();

    results[i] = string[i]; 
    numbers[i] = match;
    //atomicAdd(&count_dev, match);

   if(match != 0){
       atomicAdd(&count_dev,1);
   }

}


void search(char * string, char * results, int length) {
    char * string_dev,  *result_dev;
    int patternLength = 5;
    hipError_t err;
    int * count;
    int streamOffset;
    hipStream_t stream1;
    size_t free, free2, total;
    int * numbers; 
    int * numbers_dev;


    errorChecking( hipHostMalloc( (void**) &numbers, sizeof(int) * length , hipHostMallocDefault), __LINE__);
    for(int i = 0; i < length; ++i){
        numbers[i] = 0;
    }

    errorChecking( hipHostMalloc( (void**) &count, sizeof(int) , hipHostMallocDefault), __LINE__);
    errorChecking( hipStreamCreate(&stream1 ), __LINE__);
    *count = 0; 

    int streamLength = ceil(length/numStreams);
    int streamBytes = streamLength * sizeof(char);

    



    printf("Stream length: %d \n", streamLength);
//size     
    dim3 dimGrid((length + 1024 - 1) / 1024, 1, 1);
    dim3 dimBlock(1024, 1, 1);


    printf("dimGrid.x: %d Threads: %d \n" , dimGrid.x, dimBlock.x);

    //cudaMemGetInfo(&free, &total);
    //printf("\nFree Mem:  %zu, Total Mem: %zu \n", free, total); 


    //memset(string, 0, sizeof(char)* length);  //what does this do??
    errorChecking( hipMalloc((void **) &string_dev, sizeof(char) * length), __LINE__);
    errorChecking( hipMalloc((void **) &result_dev, sizeof(char) * length), __LINE__);
    errorChecking( hipMalloc((void **) &numbers_dev, sizeof(int) * length), __LINE__);
    errorChecking( hipMemcpyAsync(numbers_dev, numbers, sizeof(int) * length, hipMemcpyHostToDevice, stream1 ), __LINE__);
    
    errorChecking( hipMemcpyToSymbolAsync(HIP_SYMBOL(count_dev), count, sizeof(int), 0, hipMemcpyHostToDevice, stream1), __LINE__);
//size
    errorChecking( hipMemcpyAsync(string_dev, string, sizeof(char) * length, hipMemcpyHostToDevice, stream1 ), __LINE__);
    
    //cudaMemGetInfo(&free2, &total);
    //printf("Free Mem:  %zu, Total Mem: %zu \n", free2, total);
    
    search_kernel<<<dimGrid, dimBlock, 0, stream1>>>(string_dev, result_dev, numbers_dev, length);



    errorChecking(hipGetLastError(), __LINE__);
    
    errorChecking(hipMemcpyAsync(results, result_dev, sizeof(char) * length, hipMemcpyDeviceToHost, stream1), __LINE__);
    
    errorChecking(hipMemcpyFromSymbolAsync(count, HIP_SYMBOL(count_dev), sizeof(int), 0, hipMemcpyDeviceToHost, stream1), __LINE__);
    
    errorChecking(hipMemcpyAsync(numbers, numbers_dev, sizeof(int) * length, hipMemcpyDeviceToHost, stream1), __LINE__);
   
    hipStreamSynchronize(stream1); 
  
    printf("Count is: %d\n", *count);
  
    printf("Numbers\n");
    for(int i = 0; i < length; ++i){
        printf("%d ",numbers[i]);


    }

  
    errorChecking(hipStreamDestroy(stream1), __LINE__); 
    hipFree(string_dev);
    hipFree(result_dev);
    hipHostFree(count);
    hipHostFree(numbers);  
}



int main(void) {
    int length = 1000;
    int matchat = 0;
    int patternLength = 6;

    struct timeval start, end;
    char * string;
    char * results;
    errorChecking( hipHostMalloc((void**) &string, length * sizeof(char), hipHostMallocDefault), __LINE__ );
    errorChecking( hipHostMalloc((void**) &results, length * sizeof(char), hipHostMallocDefault), __LINE__ );



 
     for(int i = 0; i < length-1; ++i){
        string[i] = 'a'; 
    }   
   string[length-1] = '\0';
   
   string[4] = 'h'; 
   string[5] = 'e'; 
   string[6] = 'l'; 
   string[7] = 'l'; 
   string[8] = 'o'; 
   
   string[104] = 'h'; 
   string[105] = 'e'; 
   string[106] = 'l'; 
   string[107] = 'l'; 
   string[108] = 'o'; 


   printf("String is: %s",string);







   
    


    gettimeofday(&start, 0); 
    search(string, results, length);
    gettimeofday(&end, 0); 

    long long elapsed = (end.tv_sec-start.tv_sec)*1000000ll + end.tv_usec-start.tv_usec;
    printf("GPU Time: %lld \n", elapsed);


    results[999] = '\0';
    printf("results: %s\n", results);



    hipHostFree(string);
    hipHostFree(results);


    return 0;
}
